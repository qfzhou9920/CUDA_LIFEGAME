/*-----------------------------------------------
 * 请在此处填写你的个人信息
 * 学号:SA24218215
 * 姓名:周勤峰
 * 邮箱:qfzhou@email.ustc.edu.cn
 ------------------------------------------------*/
#include <hip/hip_runtime.h>
#include <chrono>
#include <cstring>
#include <fstream>
#include <iostream>
#include <string>

#define AT(x, y, z) universe[(x) * N * N + (y) * N + z]

// using std::cin, std::cout, std::endl;
// using std::ifstream, std::ofstream;
using namespace std;
// 存活细胞数
int population(int N, char *universe)
{
    int result = 0;
    for (int i = 0; i < N * N * N; i++)
        result += universe[i];
    return result;
}

// 打印世界状态
void print_universe(int N, char *universe)
{
    // 仅在N较小(<= 32)时用于Debug
    if (N > 32)
        return;
    for (int x = 0; x < N; x++)
    {
        for (int y = 0; y < N; y++)
        {
            for (int z = 0; z < N; z++)
            {
                if (AT(x, y, z))
                    cout << "O ";
                else
                    cout << "* ";
            }
            cout << endl;
        }
        cout << endl;
    }
    cout << "population: " << population(N, universe) << endl;
}



__global__ void life3d_kernel(int N, char *universe, char *next)
{
    // 计算当前线程处理的 (x, y, z) 坐标
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    // 确保线程在有效范围内
    if (x < N && y < N && z < N)
    {
        // 计算该细胞的邻居数量
        int alive = 0;
        for (int dx = -1; dx <= 1; dx++)
            for (int dy = -1; dy <= 1; dy++)
                for (int dz = -1; dz <= 1; dz++)
                {
                    if (dx == 0 && dy == 0 && dz == 0)
                        continue;

                    int nx = (x + dx + N) % N;
                    int ny = (y + dy + N) % N;
                    int nz = (z + dz + N) % N;
                    alive += universe[nx * N * N + ny * N + nz];
                }

        // 细胞状态更新规则
        if (universe[x * N * N + y * N + z] && (alive < 5 || alive > 7))
            next[x * N * N + y * N + z] = 0;
        else if (!universe[x * N * N + y * N + z] && alive == 6)
            next[x * N * N + y * N + z] = 1;
        else
            next[x * N * N + y * N + z] = universe[x * N * N + y * N + z];
    }
}

// 核心计算代码，利用 CUDA 将世界向前推进 T 个时刻
void life3d_run(int N, char *universe, int T)
{
    char *next = (char *)malloc(N * N * N);
    char *d_universe, *d_next;

    // 分配设备内存
    hipMalloc(&d_universe, N * N * N);
    hipMalloc(&d_next, N * N * N);

    // 将数据从主机拷贝到设备
    hipMemcpy(d_universe, universe, N * N * N, hipMemcpyHostToDevice);

    dim3 block(16, 16, 16); // 每个块的线程数
    dim3 grid((N + block.x - 1) / block.x, (N + block.y - 1) / block.y, (N + block.z - 1) / block.z); // 网格维度

    for (int t = 0; t < T; t++)
    {
        // 运行 kernel
        life3d_kernel<<<grid, block>>>(N, d_universe, d_next);

        // 等待 GPU 完成任务
        hipDeviceSynchronize();

        // 交换 universe 和 next
        char *temp = d_universe;
        d_universe = d_next;
        d_next = temp;
    }

    // 将结果从设备拷贝回主机
    hipMemcpy(universe, d_universe, N * N * N, hipMemcpyDeviceToHost);

    // 释放设备内存
    hipFree(d_universe);
    hipFree(d_next);
}


// 读取输入文件
void read_file(char *input_file, char *buffer)
{
    ifstream file(input_file, std::ios::binary | std::ios::ate);
    if (!file.is_open())
    {
        cout << "Error: Could not open file " << input_file << std::endl;
        exit(1);
    }
    std::streamsize file_size = file.tellg();
    file.seekg(0, std::ios::beg);
    if (!file.read(buffer, file_size))
    {
        std::cerr << "Error: Could not read file " << input_file << std::endl;
        exit(1);
    }
    file.close();
}

// 写入输出文件
void write_file(char *output_file, char *buffer, int N)
{
    ofstream file(output_file, std::ios::binary | std::ios::trunc);
    if (!file)
    {
        cout << "Error: Could not open file " << output_file << std::endl;
        exit(1);
    }
    file.write(buffer, N * N * N);
    file.close();
}

int main(int argc, char **argv)
{
    // cmd args
    if (argc < 5)
    {
        cout << "usage: ./life3d N T input output" << endl;
        return 1;
    }
    int N = std::stoi(argv[1]);
    int T = std::stoi(argv[2]);
    char *input_file = argv[3];
    char *output_file = argv[4];

    char *universe = (char *)malloc(N * N * N);
    read_file(input_file, universe);

    int start_pop = population(N, universe);
    auto start_time = std::chrono::high_resolution_clock::now();
    life3d_run(N, universe, T);
    auto end_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = end_time - start_time;
    int final_pop = population(N, universe);
    write_file(output_file, universe, N);

    cout << "start population: " << start_pop << endl;
    cout << "final population: " << final_pop << endl;
    double time = duration.count();
    cout << "time: " << time << "s" << endl;
    cout << "cell per sec: " << T / time * N * N * N << endl;

    free(universe);
    return 0;
}
